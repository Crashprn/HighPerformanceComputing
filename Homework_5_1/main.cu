#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>

__global__
void grey_scale_kernel(unsigned char* greyImage, unsigned char* rgbImage, int image_width, int image_height)
{
    // Calculate the row and column of the current thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Make sure we do not go outside the image bounds
    if (col < image_width && row < image_height)
    {
        int g_index = row * image_width + col;
        int rgb_index = g_index * 3;
        // Calculate the grey value
        greyImage[g_index] = 0.21 * rgbImage[rgb_index] + 0.72 * rgbImage[rgb_index + 1] + 0.07 * rgbImage[rgb_index + 2];
    }
}

__host__
void read_image_file(std::string filename, char* buffer, int size)
{
    std::ifstream file(filename, std::ios::binary);
    if (file.is_open())
    {
        file.read(buffer, size);
        file.close();
    }
    else
    {
        std::cout << "Unable to open file" << std::endl;
    }
}

void write_image_file(std::string filename, char* buffer, int size)
{
    std::ofstream file(filename, std::ios::binary);
    if (file.is_open())
    {
        file.write(buffer, size);
        file.close();
    }
    else
    {
        std::cout << "Unable to open file" << std::endl;
    }
}

int main()
{
    // Set image properties
    int image_width = 1024;
    int image_height = 1024;
    int image_channels = 3;
    int image_size = image_width * image_height * image_channels;

    // Read image file
    char* rgb_image = new char[image_size];
    read_image_file("gc_conv_1024x1024.raw", rgb_image, image_size);

    // Allocate memory for grey image
    char* grey_image = new char[image_width * image_height];

    // Create device pointers
    unsigned char* d_rgb_image, *d_grey_image;

    // Copy data to device
    hipMalloc((void**)&d_rgb_image, image_size * sizeof(unsigned char));
    hipMemcpy(d_rgb_image, rgb_image, image_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_grey_image, image_width * image_height * sizeof(unsigned char));
    hipMemcpy(d_grey_image, grey_image, image_width * image_height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 DimGrid(ceil(image_width / 16.0), ceil(image_height/ 16.0), 1);
    dim3 DimBlock(16, 16, 1);

    // Launch kernel
    grey_scale_kernel<<<DimGrid, DimBlock>>>(d_grey_image, d_rgb_image, image_width, image_height);

    // Copy data back to host
    hipDeviceSynchronize();
    hipMemcpy(grey_image, d_grey_image, image_width * image_height * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipFree(d_rgb_image);
    hipFree(d_grey_image);

    // Write grey image file
    write_image_file("gc_conv_1024x1024_grey.raw", grey_image, image_width * image_height);
}